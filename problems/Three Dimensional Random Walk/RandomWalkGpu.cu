#include "hip/hip_runtime.h"
#include "Random.cu"

// Number of threads per block.
#define noOfThreads 1024

// Variable in global memory to store total distance of all particles.
__device__ double totalDistance;

// Per-thread variables in shared memory.
__shared__ long long int x [noOfThreads];
__shared__ long long int y [noOfThreads];
__shared__ long long int z [noOfThreads];

/**
 * Atomically set double variable v to the sum of itself and value.
 *
 * @param  v      Pointer to double variable.
 * @param  value  Value.
 */
__device__ void atomicAdd(double *v, double value) {
	double oldval, newval;
	do {
		oldval = *v;
		newval = oldval + value;
	} while (atomicCAS((unsigned long long int *)v,__double_as_longlong (oldval), __double_as_longlong (newval)) != __double_as_longlong (oldval));
}

/**
 * Device kernel to compute total distance of particles.
 * <P>
 * Called with a one-dimensional grid of one-dimensional blocks.
 *
 * @param  noOfSteps	Number of steps.
 * @param  seed			Pseudorandom number generator seed.
 *
 * @author Umang Jethalal Gala
 * @version 11-Nov-2014
 */
extern "C" __global__ void calculateDistance(unsigned long long int noOfSteps, long long int seed) {
	int thrdIndex;
	int rank;
	prng_t prng;
	long long int thrdX = 0;
	long long int thrdY = 0;
	long long int thrdZ = 0;
	
	// Determine the current thread Index.
	thrdIndex = threadIdx.x;
	rank = blockIdx.x * noOfThreads + thrdIndex;
	
	// Initialize per-thread prng.
	prngSetSeed (&prng, seed + rank);
	
	// Compute steps for particles
	for (unsigned long long int step = thrdIndex; step < noOfSteps; step += noOfThreads) {
		int choice = prngNextInt (&prng, 6);
		if (choice == 0) {
			thrdX -= 1;
		} else if (choice == 1) {
			thrdX += 1;
		} else if (choice == 2) {
			thrdY -= 1;
		} else if (choice == 3) {
			thrdY += 1;
		} else if (choice == 4) {
			thrdZ -= 1;
		} else if (choice == 5) {
			thrdZ += 1;
		}
	}
	
	// Shared memory parallel reduction within thread block.
	x[thrdIndex] = thrdX;
	y[thrdIndex] = thrdY;
	z[thrdIndex] = thrdZ;
	__syncthreads();
	for (int th = noOfThreads/2; th > 0; th >>=1) {
		if (thrdIndex < th) {
			x[thrdIndex] += x[thrdIndex + th];
			y[thrdIndex] += y[thrdIndex + th];
			z[thrdIndex] += z[thrdIndex + th];
		}
		__syncthreads();
	}
	
	// Calculate distance for each particle and perform atomic addition to determine total distance.
	if (thrdIndex == 0) {
		double distance = sqrt((double)(x[0]*x[0]) + (y[0]*y[0]) + (z[0]*z[0]));
		atomicAdd (&totalDistance, distance);
	}
}
